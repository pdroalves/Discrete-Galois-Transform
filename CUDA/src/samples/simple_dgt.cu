// cuPoly - A GPGPU-based library for doing polynomial arithmetic on RLWE-based cryptosystems
// Copyright (C) 2017, Pedro G. M. R. Alves - pedro.alves@ic.unicamp.br

// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#include <dgt/dgt.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

 int main(int argc, char* argv[]){

  const int n = 4096;
  const int nresidues = 5; // 5 * 64 = 320 bits?

  // Setup
  set_const_mem(n/2);
  GaloisInteger *d_data;
  hipMalloc((void**)&d_data,nresidues * (n/2) * sizeof(GaloisInteger));
  cudaCheckError();
  
  hipProfilerStart();
  execute_dgt(d_data, n, nresidues, FORWARD);
  execute_dgt(d_data, n, nresidues, INVERSE);
  hipProfilerStop();

  hipFree(d_data);
}
